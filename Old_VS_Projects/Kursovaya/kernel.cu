#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <math.h>
#include <iomanip>
#include <ctime>
#include <chrono> 
using namespace std;

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Ошибка CUDA: %s\n", hipGetErrorString(err));    \
printf("В файле: %s, на строке: %i\n", __FILE__, __LINE__);  \
}                 \

#define num 5000 // num - количество корней 20000
#define threads_ 1024 // threads_ - количество нитей в блоке 1024


__global__ void Create(double* Array, unsigned long int N)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N * N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	if (q < w) {
		Array[ind] = 0;
	}
	else {
		double tmp = 1 + 0.2 * ind;
		Array[ind] = tmp;
	}
}

__global__ void fillone(double* Array)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > num) { return; }
	Array[ind] = 1.0f;
}

__global__ void ParalelK1(double* L, double* B, double* X, unsigned long int N, unsigned long int j)
{
	unsigned long int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind == 0) {
		X[j] = B[j] / L[j * N + j];
	}
}

__global__ void ParalelK2(double* L, double* B, double* X, unsigned long int N, unsigned long int j, unsigned long int i)
{
	unsigned long int ind = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long int w = (ind % N);
	unsigned long int q = (ind / N);
	__shared__ float tmpX;
	__shared__ unsigned long int T;
	if (threadIdx.x == 0) {
		tmpX = X[j];
		T = N;
	}
	__syncthreads();
	if (w == i) {
		B[q] = B[q] - L[q * T + j] * tmpX;
	}
	__syncthreads();
}

void ParalelSolve(double* L, double* B, double* X, unsigned long int N)
{
	dim3 threads = dim3(threads_, 1);
	dim3 blocks = dim3((N * N / threads.x) + 1, 1);
	for (unsigned long int j = 0; j < N; j++) {
		ParalelK1 << < 1, 1 >> > (L, B, X, N, j);
		unsigned long int i = j + 1;
		ParalelK2 << < blocks, threads >> > (L, B, X, N, j, i);
	}
}

__global__ void ParalelSolveK(double* L, double* B, double* X, unsigned long int N)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	__shared__ float tmpX;
	__shared__ float tmpB;
	for (unsigned long int j = 0; j < N; j++) {
		if (threadIdx.x == 0) {
			tmpX = X[j];
			tmpB = B[j];
		}
		__syncthreads();
		if (ind == 0) { tmpX = tmpB / L[j * N + j]; }
		__syncthreads();
		if (threadIdx.x == 0) {
			X[j] = tmpX;
		}
		unsigned long int i = j + 1;
		__syncthreads();
		if (w == i) { B[q] = B[q] - L[q * N + j] * tmpX; }
		__syncthreads();
	}
}

void LinelSolve(double* L, double* B, double* X, unsigned long int N)
{
	for (unsigned long int j = 0; j < N; j++) {
		X[j] = B[j] / L[j * N + j];
		for (unsigned long int i = j + 1; i < N; i++) {
			B[i] = B[i] - L[i * num + j] * X[j];
		}
	}
}

int main(void)
{
	setlocale(0, "");
	hipEvent_t start, stop;
	unsigned long int j = 0, n = num;
	double* a = (double*)malloc(n * n * sizeof(*a));    // nxn matrix a on the host       
	double* y = (double*)malloc(n * sizeof(*y));
	double* O = (double*)malloc(n * n * sizeof(*O));
	unsigned int* P = (unsigned int*)malloc(sizeof(*P));
	float gpuTime = 0;

	double* d_a = NULL; // d_a - a on the device
	double* d_y = NULL;
	double* d_O = NULL;

	CUDA_CHECK_ERROR(hipMalloc((void**)&d_a, n * n * sizeof(*a))); // device memory alloc for a
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_y, n * sizeof(*y)));
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_O, n * n * sizeof(*O)));

	dim3 threads = dim3(threads_, 1);
	dim3 blocks = dim3((n * n / threads.x) + 1, 1);
	Create << < blocks, threads >> > (d_a, n);

	blocks = dim3((n / threads.x) + 1, 1);
	fillone << < blocks, threads >> > (d_y);

	/*CUDA_CHECK_ERROR(hipMemcpy(a, d_a, n * n * sizeof(*a), hipMemcpyDeviceToHost));
	cout << "Матрица a: \n";
	for (j = 0; j < n * n; j++) {
		cout << fixed << setprecision(3) << a[j] << " ";
		if ((j % n) == (n - 1)) cout << endl;
	}
	cout << "\n";*/

	CUDA_CHECK_ERROR(hipDeviceSynchronize());

	CUDA_CHECK_ERROR(hipMemcpy(a, d_a, n * n * sizeof(*a), hipMemcpyDeviceToHost));
	CUDA_CHECK_ERROR(hipMemcpy(y, d_y, n * sizeof(*y), hipMemcpyDeviceToHost));

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	CUDA_CHECK_ERROR(hipEventRecord(start, 0));

	//blocks = dim3((n * n / threads.x) + 1, 1);

	//ParalelSolveK << < blocks, threads >> > (d_a, d_y, d_O, n);

	ParalelSolve (d_a, d_y, d_O, n);

	CUDA_CHECK_ERROR(hipEventRecord(stop, 0));
	CUDA_CHECK_ERROR(hipEventSynchronize(stop));
	CUDA_CHECK_ERROR(hipEventElapsedTime(&gpuTime, start, stop));
	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	CUDA_CHECK_ERROR(hipDeviceSynchronize());

	cout << setprecision(6) << "Для вычисления понадобилось " << gpuTime << " милисекунд\n";

	/*CUDA_CHECK_ERROR(hipMemcpy(O, d_O, n * sizeof(*O), hipMemcpyDeviceToHost));
	printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << O[j] << endl;
	}*/

	/*CUDA_CHECK_ERROR(hipMemcpy(O, d_y, n * sizeof(*O), hipMemcpyDeviceToHost));
	printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << O[j] << endl;
	}*/

	auto start_time = chrono::steady_clock::now();

	//LinelSolve(a, y, O, n);

	auto end_time = chrono::steady_clock::now();
	auto elapsed_ns = chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time);
	//cout << setprecision(6) << "Для вычисления понадобилось " << elapsed_ns.count() * 0.000001 << " милисекунд\n";

	/*printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << O[j] << endl;
	}*/

	/*printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << y[j] << endl;
	}*/

	CUDA_CHECK_ERROR(hipFree(d_a));               // free device memory
	CUDA_CHECK_ERROR(hipFree(d_y));               // free device memory 
	CUDA_CHECK_ERROR(hipFree(d_O));
	free(a);                                       // freе host memory
	free(y);                                       // free host memory
	free(O);
	cin.get();

	return EXIT_SUCCESS;
}