#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <math.h>
#include <iomanip>
#include "hipblas.h"
#pragma comment (lib, "cublas.lib")
using namespace std;

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Ошибка CUDA: %s\n", hipGetErrorString(err));    \
printf("В файле: %s, на строке: %i\n", __FILE__, __LINE__);  \
}                 \

#define CUBLAS_CHECK_ERROR(err)           \
if (err != HIPBLAS_STATUS_SUCCESS) {          \
printf("Ошибка cuBLAS: %s\n", err);    \
printf("В файле: %s, на строке: %i\n", __FILE__, __LINE__);  \
}                 \

#define num 5000 // num - количество корней 20000

__global__ void Create(double* Array, unsigned long int N)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N * N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	if (q > w) {
		Array[ind] = 0;
	}
	else {
		double tmp = 1 + 0.2 * ind;
		Array[ind] = tmp;
	}
}

__global__ void fillone(double* Array)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > num) { return; }
	Array[ind] = 1.0f;
}

int main(void)
{
	setlocale(0, "");
	hipEvent_t start, stop;
	hipblasHandle_t handle;                              // CUBLAS context
	unsigned long int j, n = num;
	double* a = (double*)malloc(n * n * sizeof(*a));    // nxn matrix a on the host
	double* x = (double*)malloc(n * sizeof(*x));        // n - vector x on the host
	float gpuTime = 0;

	double* d_a = NULL; // d_a - a on the device
	double* d_x = NULL; // d_x - x on the device

	CUDA_CHECK_ERROR(hipMalloc((void**)&d_a, n * n * sizeof(*a))); // device memory alloc for a
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_x, n * sizeof(*x)));     // device memory alloc for x
	CUBLAS_CHECK_ERROR(hipblasCreate(&handle));                   // initialize CUBLAS context

	dim3 threads = dim3(1024, 1);
	dim3 blocks = dim3((n * n / threads.x) + 1, 1);
	Create << < blocks, threads >> > (d_a, n);

	blocks = dim3((n / threads.x) + 1, 1);
	fillone << < blocks, threads >> > (d_x);

	/*CUDA_CHECK_ERROR(hipMemcpy(a, d_a, n * n * sizeof(*a), hipMemcpyDeviceToHost));
	cout << "Матрица a: \n";
	for (int h = 0; h < n * n; h++) {
		cout << fixed << setprecision(3) << a[h] << " ";
		if ((h % n) == (n - 1)) cout << endl;
	}
	cout << "\n"; */

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	CUDA_CHECK_ERROR(hipEventRecord(start, 0));

	CUBLAS_CHECK_ERROR(hipblasDtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, d_a, n, d_x, 1));

	CUDA_CHECK_ERROR(hipEventRecord(stop, 0));
	CUDA_CHECK_ERROR(hipEventSynchronize(stop));
	CUDA_CHECK_ERROR(hipEventElapsedTime(&gpuTime, start, stop));
	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	cout << setprecision(6) << "Для вычисления понадобилось " << gpuTime << " милисекунд\n";

	CUDA_CHECK_ERROR(hipDeviceSynchronize());

	/*CUDA_CHECK_ERROR(hipMemcpy(x, d_x, n * sizeof(*x), hipMemcpyDeviceToHost));
	printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << x[j] << endl;
	}
	cout << endl;*/

	CUDA_CHECK_ERROR(hipFree(d_a));               // free device memory
	CUDA_CHECK_ERROR(hipFree(d_x));               // free device memory 
	CUBLAS_CHECK_ERROR(hipblasDestroy(handle));  // destroy CUBLAS context
	free(a);                                       // freе host memory
	free(x);                                       // free host memory
	cin.get();
	return EXIT_SUCCESS;
}




























































