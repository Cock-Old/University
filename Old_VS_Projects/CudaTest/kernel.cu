#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <math.h>
#include <iomanip>
#include "hipblas.h"
#pragma comment (lib, "cublas.lib")
#include <ctime>
#include <chrono> 
using namespace std;

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Ошибка CUDA: %s\n", hipGetErrorString(err));    \
printf("В файле: %s, на строке: %i\n", __FILE__, __LINE__);  \
}                 \

#define CUBLAS_CHECK_ERROR(err)           \
if (err != HIPBLAS_STATUS_SUCCESS) {          \
printf("Ошибка cuBLAS: %s\n", err);    \
printf("В файле: %s, на строке: %i\n", __FILE__, __LINE__);  \
}                 \

#define num 10 // num - количество корней 20000

__global__ void Create(double* Array, unsigned long int N)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N * N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	if (q > w) {
		Array[ind] = 0;
	}
	else {
		double tmp = 1 + 0.2 * ind;
		Array[ind] = tmp;
	}
}

__global__ void Transpose(double* Array, unsigned long int N)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N * N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	if (q < w) {
		Array[w * N + q] = Array[ind] - Array[w * N + q];
		Array[ind] = 0;
	}
}

__global__ void fillone(double* Array)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > num) { return; }
	Array[ind] = 1.0f;
}

/*__global__ void ParalelSolve(double* L, double* B, double* X, unsigned long int N)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N * N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	for (unsigned long int j = 0; j < N; j++) {
		if (w == j) { X[j] = B[j] / L[j * N + j]; }
		__syncthreads();
		for (unsigned long int i = j + 1; i < N; i++) {
			if ((q == i) && (w == j)) { B[i] = B[i] - L[i * num + j] * X[j]; }
		}
		__syncthreads(); 
	}
}*/

__global__ void Paralel1(double* L, double* B, double* X, unsigned long int N, unsigned long int j)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	if (w == j) { X[j] = B[j] / L[j * N + j]; }
	__syncthreads();
}

__global__ void Paralel2(double* L, double* B, double* X, unsigned long int N, unsigned long int j, unsigned long int i)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind > N * N) { return; }
	unsigned long int q = (ind / N);
	unsigned long int w = (ind % N);
	if ((q == i) && (w == j)) { B[i] = B[i] - L[i * num + j] * X[j]; }
	__syncthreads();
}

void ParalelSolve(double* L, double* B, double* X, unsigned long int N)
{
	dim3 threads = dim3(1024, 1);
	dim3 blocks = dim3((N * N/ threads.x) + 1, 1);
	for (unsigned long int j = 0; j < N; j++) {
		Paralel1 << < blocks, threads >> > (L, B, X, N, j);
		for (unsigned long int i = j + 1; i < N; i++) {
			Paralel2 << < blocks, threads >> > (L, B, X, N, j, i);
		}
	}
}

void LinelSolve(double* L, double* B, double* X, unsigned long int N)
{
	for (unsigned long int j = 0; j < N; j++) {
		X[j] = B[j] / L[j * N + j];
		for (unsigned long int i = j + 1; i < N; i++) {
			B[i] = B[i] - L[i * num + j] * X[j];
		}
	}
}

int main(void)
{
	setlocale(0, "");
	hipEvent_t start, stop;
	hipblasHandle_t handle;                              // CUBLAS context
	unsigned long int j, n = num;
	double* a = (double*)malloc(n * n * sizeof(*a));    // nxn matrix a on the host
	double* x = (double*)malloc(n * sizeof(*x));        // n - vector x on the host
	double* y = (double*)malloc(n * sizeof(*y));
	double* O = (double*)malloc(n * n * sizeof(*O));
	float gpuTime = 0;

	double* d_a = NULL; // d_a - a on the device
	double* d_x = NULL; // d_x - x on the device
	double* d_y = NULL;
	double* d_O = NULL;

	CUDA_CHECK_ERROR(hipMalloc((void**)&d_a, n * n * sizeof(*a))); // device memory alloc for a
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_x, n * sizeof(*x)));     // device memory alloc for x
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_y, n * sizeof(*y)));
	CUDA_CHECK_ERROR(hipMalloc((void**)&d_O, n * n * sizeof(*O)));
	CUBLAS_CHECK_ERROR(hipblasCreate(&handle));                   // initialize CUBLAS context


	dim3 threads = dim3(1024, 1);
	dim3 blocks = dim3((n * n / threads.x) + 1, 1);
	Create << < blocks, threads >> > (d_a, n);

	blocks = dim3((n / threads.x) + 1, 1);
	fillone << < blocks, threads >> > (d_x);

	/*CUDA_CHECK_ERROR(hipMemcpy(a, d_a, n * n * sizeof(*a), hipMemcpyDeviceToHost));
	cout << "Матрица a: \n";
	for (int h = 0; h < n * n; h++) {
		cout << fixed << setprecision(3) << a[h] << " ";
		if ((h % n) == (n - 1)) cout << endl;
	}
	cout << "\n"; */

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	CUDA_CHECK_ERROR(hipEventRecord(start, 0));

	CUBLAS_CHECK_ERROR(hipblasDtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, d_a, n, d_x, 1));

	CUDA_CHECK_ERROR(hipEventRecord(stop, 0));
	CUDA_CHECK_ERROR(hipEventSynchronize(stop));
	CUDA_CHECK_ERROR(hipEventElapsedTime(&gpuTime, start, stop));
	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	cout << setprecision(6) << "Для вычисления понадобилось " << gpuTime << " милисекунд\n";

	CUDA_CHECK_ERROR(hipDeviceSynchronize());

	CUDA_CHECK_ERROR(hipMemcpy(x, d_x, n * sizeof(*x), hipMemcpyDeviceToHost));
	printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << x[j] << endl;
	}
	cout << endl;

	threads = dim3(1024, 1);
	blocks = dim3((n * n / threads.x) + 1, 1);
	Transpose << < blocks, threads >> > (d_a, n);

	blocks = dim3((n / threads.x) + 1, 1);
	fillone << < blocks, threads >> > (d_y);

	CUDA_CHECK_ERROR(hipDeviceSynchronize());

	CUDA_CHECK_ERROR(hipMemcpy(a, d_a, n * n * sizeof(*a), hipMemcpyDeviceToHost));
	CUDA_CHECK_ERROR(hipMemcpy(y, d_y, n * sizeof(*y), hipMemcpyDeviceToHost));

	/*cout << "Матрица a: \n";
	for (int h = 0; h < n * n; h++) {
	  cout << setprecision(3) << a[h] << " ";
	  if ((h % n) == (n - 1)) cout << endl;
	}
	cout << "\n"; */

	threads = dim3(1024, 1);
	blocks = dim3((n * n / threads.x) + 1, 1);

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	CUDA_CHECK_ERROR(hipEventRecord(start, 0));

	//ParalelSolve (d_a, d_y, d_O, n);

	CUDA_CHECK_ERROR(hipEventRecord(stop, 0));
	CUDA_CHECK_ERROR(hipEventSynchronize(stop));
	CUDA_CHECK_ERROR(hipEventElapsedTime(&gpuTime, start, stop));
	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	CUDA_CHECK_ERROR(hipDeviceSynchronize());

	cout << setprecision(6) << "Для вычисления понадобилось " << gpuTime << " милисекунд\n";

	CUDA_CHECK_ERROR(hipMemcpy(O, d_O, n * sizeof(*O), hipMemcpyDeviceToHost));
	printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << O[j] << endl;
	}

	auto start_time = chrono::steady_clock::now();

	LinelSolve(a, y, O, n);

	auto end_time = chrono::steady_clock::now();
	auto elapsed_ns = chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time);
	cout << setprecision(6) << "Для вычисления понадобилось " << elapsed_ns.count()* 0.000001 << " милисекунд\n";

	/*printf("Решение:\n");
	for (j = 0; j < n; j++) {
		cout << j << " " << setprecision(6) << O[j] << endl;
	}*/

	CUDA_CHECK_ERROR(hipFree(d_a));               // free device memory
	CUDA_CHECK_ERROR(hipFree(d_x));               // free device memory 
	CUBLAS_CHECK_ERROR(hipblasDestroy(handle));                      // destroy CUBLAS context
	free(a);                                       // freе host memory
	free(x);                                       // free host memory

	return EXIT_SUCCESS;
}