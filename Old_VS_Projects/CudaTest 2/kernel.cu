#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>

#define N 10

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int id = threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void generate(hiprandState* globalState, float* randomArray)
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    randomArray[ind] = RANDOM;
    globalState[ind] = localState;
}

int main(int argc, char** argv)
{
    dim3 tpb(N, 1);
    hiprandState* devStates;
    float* randomValues = new float[N];
    float* devRandomValues;

    hipMalloc(&devStates, N * sizeof(hiprandState));
    hipMalloc(&devRandomValues, N * sizeof(*randomValues));// setup seeds
    setup_kernel << <1, N >> > (devStates, time(NULL));

    printf("%s\n", hipGetErrorString(hipGetLastError()));// generate random numbers
    generate << <1, N >> > (devStates, devRandomValues);

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    hipMemcpy(randomValues, devRandomValues, N * sizeof(*randomValues), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%f\n", randomValues[i]);
    }

    hipFree(devRandomValues);
    hipFree(devStates);
    delete randomValues;
    getchar();
    return 0;
}